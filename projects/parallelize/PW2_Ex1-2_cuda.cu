#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

#include "hip/hip_vector_types.h"

__device__ float Gaussian(float x, float y, float sigma) {
    return expf(-(x * x + y * y) / (2.0f * sigma * sigma)) / (2.0f * 3.14159265f * sigma * sigma);
}

__device__ float3 Gaussian_conv(const cv::cuda::PtrStep<float3> source, int cols, int rows, int i, int j, float kSize, float sigma)
{
    int halfSize = static_cast<int>(kSize / 2.0f);
    float3 colorSum = make_float3(0.0f, 0.0f, 0.0f);
    float weightSum = 0.0f;
    int halfWidth = cols / 2;
    float3 pixelValue;

    for (int dy = -halfSize; dy <= halfSize; ++dy) {
        for (int dx = -halfSize; dx <= halfSize; ++dx) {
            int y = i + dy;
            int x = j + dx;

            if (x < 0) {
                pixelValue = source(y, abs(x - 1));
            } else if (x > cols) {
                pixelValue = source(y, cols - x + 1);
            } else if (y < 0) {
                pixelValue = source(abs(y - 1), x);
            } else if (y > rows) {
                pixelValue = source(rows - y + 1, x);
            } else if (j <= halfWidth) {
                if (halfWidth - x >= 0) {
                    pixelValue = source(y, x);
                } else {
                    pixelValue = source(y, halfWidth - x + 1 + halfWidth);
                }
            } else if (j > halfWidth) {
                if (x - halfWidth > 0) {
                    pixelValue = source(y, x);
                } else {
                    pixelValue = source(y, halfWidth - x + 1 + halfWidth);
                }
            } else {
                pixelValue = source(y, x);
            }

            // Clamp coordinates to image borders
            // y = max(0, min(y, rows - 1));
            // x = max(0, min(x, cols - 1));

            // float3 pixel = source(y, x);

            float weight = Gaussian(dx, dy, sigma);
            weightSum += weight;
            // colorSum += weight * pixel;
            colorSum += weight * pixelValue;
        }
    }

    // Normalize
    colorSum /= weightSum;

    return colorSum;
}

__global__ void process(const cv::cuda::PtrStep<float3> src,
                        cv::cuda::PtrStep<float3> dst,
                        int rows, int cols, float kSize, float sigma) 
{

    const int j = blockDim.x * blockIdx.x + threadIdx.x;
    const int i = blockDim.y * blockIdx.y + threadIdx.y;

    if ((j > 0) && (j < cols - 1) && (i < rows - 1) && (i > 0)) // Ensure the coordinate is in source
    {
        float3 resultPixel;
        
        resultPixel = Gaussian_conv(src, cols, rows, i, j, kSize, sigma);

        // clamp(resultPixel, 0.0, 1.0);
        resultPixel.x = fminf(fmaxf(resultPixel.x, 0.0f), 1.0f);
        resultPixel.y = fminf(fmaxf(resultPixel.y, 0.0f), 1.0f);
        resultPixel.z = fminf(fmaxf(resultPixel.z, 0.0f), 1.0f);
        dst(i, j) = resultPixel;
    }
}

int divUp(int a, int b) // Ensures CUDA grid dimensions are big enough.
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void startCUDA(cv::cuda::GpuMat &src, cv::cuda::GpuMat &dst, float kSize, float sigma)
{
    const dim3 block(32, 8);
    const dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));

    process<<<grid, block>>>(src, dst, src.rows, src.cols, kSize, sigma);
}
